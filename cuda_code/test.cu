#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"

void test(double* input, int n, int p) {
    double* device_input;

    hipMalloc((void **)&device_input, sizeof(double) * n * p);
    hipMemcpy(device_input, input, sizeof(double) * n * p, hipMemcpyHostToDevice);
    hipFree(device_input);

}
