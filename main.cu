#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <time.h>

extern "C"{

#include "external/data.h"
#include "external/tree.h"
#include "external/eval.h"
#include "external/forest.h"

const DecisionTreeNode **train_model_parallel(double **data,
    const RandomForestParameters *params,
    const struct dim *csv_dim,
    const ModelContext *ctx,
    unsigned int num_threads);

double validate(
    double **data,
    const RandomForestParameters *params,
    const struct dim *csv_dim,
    const unsigned int procnum)
{
    const ModelContext ctx = (const ModelContext){
        testingFoldIdx : 0 /* Fold to use for evaluation. */,
        rowsPerFold : csv_dim->rows / 1 /* Number of rows per fold. */
    };
    const DecisionTreeNode **random_forest = (const DecisionTreeNode **)train_model_parallel(
        data, params, csv_dim, &ctx, procnum);
    
    //const DecisionTreeNode **random_forest = (const DecisionTreeNode **)train_model(
    //    data, params, csv_dim, &ctx);

    double accuracy = eval_model(
        random_forest, data, params, csv_dim, &ctx);
    
    free_random_forest(&random_forest, params->n_estimators);

    return accuracy;
}

int main(int argc, char **argv){
    int num_threads = 10;
    size_t num_trees = 1000;
    // === Random seed ===
    //srand(0);
    srand(time(NULL));

    // === Read and parse DATA ===
    const char *file_name = "heart.csv";
    struct dim csv_dim;
    csv_dim = parse_csv_dims(file_name);
    // Allocate memory for the data coming from the .csv and read in the data.
    double *data = (double*)malloc(sizeof(double) * csv_dim.rows * csv_dim.cols);
    parse_csv(file_name, &data, csv_dim);

    // === Config random forest model ===
    const RandomForestParameters params = {
        n_estimators : num_trees /* Number of trees in the random forest model. */,
        max_depth : 7 /* Maximum depth of a tree in the model. */,
        min_samples_leaf : 3,
        max_features : 3
    };

    // Pivot the csv file data into a two dimensional array.
    double **pivoted_data;
    pivot_data(data, csv_dim, &pivoted_data);

    // === Train and validation ===
    clock_t begin_clock = clock();
    double accuracy = validate(pivoted_data, &params, &csv_dim, num_threads);
    clock_t end_clock = clock();

    printf("cross validation accuracy: %f%% (%ld%%)\n", (accuracy * 100), (long)(accuracy * 100));
    printf("(time taken: %fs)\n", (double)(end_clock - begin_clock) / CLOCKS_PER_SEC);

    // Free loaded csv file data.
    free(data);
    free(pivoted_data);
}
}